#include "hip/hip_runtime.h"

#include "GPUUtils.hpp"

using namespace cv;
using namespace cv::cuda;

// Huge props to wykvictor for this solution https://github.com/opencv/opencv/issues/6295#issuecomment-246647886

__global__ void inRangeCudaKernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst, int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x >= src.cols || y >= src.rows) return;

	uchar3 v = src(y, x);
	if(v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
		dst(y, x) = 255;
	else
		dst(y, x) = 0;
}

void inRangeGPU(cv::InputArray _src, cv::Scalar &lowerb, cv::Scalar &upperb, cv::OutputArray _dst) {
	const int m = 32;

	GpuMat src = _src.getGpuMat();
	const int depth = _src.depth();
	int numRows = src.rows, numCols = src.cols;

	CV_Assert( depth == CV_8U );
	CV_Assert( src.channels() == 3 );
	CV_Assert( numRows > 0 );
	CV_Assert( numCols > 0 );

	_dst.create(_src.size(), CV_8UC1);
	GpuMat dst = _dst.getGpuMat();

	// Attention! Cols Vs. Rows are reversed
	const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
	const dim3 blockSize(m, m, 1);

	inRangeCudaKernel<<<gridSize, blockSize>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1], lowerb[2], upperb[2]);
}
